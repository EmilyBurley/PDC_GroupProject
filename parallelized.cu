#include <stdio.h>
#include <hip/hip_runtime.h>

__global__ void matrixMulGlobalKernel(int* pfMatrixA, int* pfMatrixB, int* pfMatrixC, int m, int n, int k) {
    int nRow = blockIdx.y * blockDim.y + threadIdx.y;
    int nCol = blockIdx.x * blockDim.x + threadIdx.x;
    int fCVal = 0;

    if (nRow < m && nCol < n) {
        for (int i = 0; i < k; i++) {
            fCVal += pfMatrixA[nRow * k + i] * pfMatrixB[i * n + nCol];
        }

        pfMatrixC[nRow * n + nCol] = fCVal;
    }
}

int main() {
    // Number of rows in Matrix 1 and Matrix 3.
    const int M = 1000;
    // Number of columns in Matrix 2 and Matrix 3.
    const int N = 1000;
    // Number of columns in Matrix 1 and rows in Matrix 2.
    const int K = 1000;

    int* h_A = (int*)malloc(M * K * sizeof(int));
    int* h_B = (int*)malloc(K * N * sizeof(int));
    int* h_C = (int*)malloc(M * N * sizeof(int));

    for (int i = 0; i < M * K; i++) {
        h_A[i] = 7;
    }

    for (int i = 0; i < K * N; i++) {
        h_B[i] = 8;
    }

    int* d_A, * d_B, * d_C;
    hipMalloc(&d_A, M * K * sizeof(int));
    hipMalloc(&d_B, K * N * sizeof(int));
    hipMalloc(&d_C, M * N * sizeof(int));

    hipMemcpy(d_A, h_A, M * K * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, K * N * sizeof(int), hipMemcpyHostToDevice);

    dim3 blockDim(16, 16);
    dim3 gridDim((N + blockDim.x - 1) / blockDim.x, (M + blockDim.y - 1) / blockDim.y);

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    hipEventRecord(start);

    matrixMulGlobalKernel<<<gridDim, blockDim>>>(d_A, d_B, d_C, M, N, K);

    hipEventRecord(stop);
    hipEventSynchronize(stop);

    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);

    hipMemcpy(h_C, d_C, M * N * sizeof(int), hipMemcpyDeviceToHost);

    /*
    printf("Result Matrix:\n");
    for (int i = 0; i < M; i++) {
        for (int j = 0; j < N; j++) {
            printf("%d ", h_C[i * N + j]);
        }
        printf("\n");
    }
    */

    printf("Global Memory Kernel Elapsed Time: %f milliseconds\n", milliseconds);

    free(h_A);
    free(h_B);
    free(h_C);
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);

    hipEventDestroy(start);
    hipEventDestroy(stop);

    return 0;
}
